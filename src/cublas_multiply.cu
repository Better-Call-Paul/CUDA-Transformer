#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <iostream>
#include <cassert>

/*
 * Assumed to be in column major form
*/
void verify_solution(float *a, float *b, float *c, int n, const float epsilon) {
    float temp_sum;
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            temp_sum = 0.0f;
            for (int k = 0; k < n; k++) {
                temp_sum += a[k * n + i] * b[j * n + k];
            }
            assert(std::fabs(c[j * n + i] - temp_sum) < epsilon);
        }
    }
    std::cout << "Cublas Kernel Successful" << "\n";
}

int main() {

    const float epsilon = 1e-3;
    int n = 1 << 10;
    size_t size = n * n * sizeof(float);


    float *host_vector_a, *host_vector_b, *host_vector_c;
    float *device_vector_a, *device_vector_b, *device_vector_c;

    host_vector_a = (float*)malloc(size);
    host_vector_b = (float*)malloc(size);
    host_vector_c = (float*)malloc(size);
    hipMalloc(&device_vector_a, size);
    hipMalloc(&device_vector_b, size);
    hipMalloc(&device_vector_c, size);

    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long)clock());

    // Fill matrixes on device
    hiprandGenerateUniform(prng, device_vector_a, n * n);
    hiprandGenerateUniform(prng, device_vector_b, n * n);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float alpha = 1.0f;
    float beta = 0.0f;


    // c = (alpha * a) * b + (beta * c)
    // (m x n) * (n x k) = (m x k)
    // Signature: handle, operation, operation, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc  : ld is leading dimension of x
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, device_vector_a, n, device_vector_b, n, &beta, device_vector_c, n); // CUBLAS_OP_N is reg matrix, OP_T is transpose

    hipMemcpy(host_vector_a, device_vector_a, size, hipMemcpyDeviceToHost);
    hipMemcpy(host_vector_b, device_vector_b, size, hipMemcpyDeviceToHost);
    hipMemcpy(host_vector_c, device_vector_c, size, hipMemcpyDeviceToHost);

    verify_solution(host_vector_a, host_vector_b, host_vector_c, n, epsilon);

    free(host_vector_a);
    free(host_vector_b);
    free(host_vector_c);

    hipFree(device_vector_a);
    hipFree(device_vector_b);
    hipFree(device_vector_c);

    return 0;
}

