#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <iostream>

__global__ float mean() {

}

__global__ float variance() {

}


__global__ void layer_norm() {

}


int main() {

    int number = 1 << 8;
    int N = number, M = number;
    size_t size = N * M * sizeof(float);

    float *host_input_vector, *host_output_vector, *device_input_vector, *device_output_vector;

    host_input_vector = (float*)malloc(size);
    host_output_vector = (float*)malloc(size);
    
    // Populate host vector
    hiprandGenerator_t generator;
    hiprandCreateGeneratorHost(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(generator, (unsigned long long)clock());
    hiprandGenerateUniform(generator, host_input_vector, N * M);

    hipMalloc(&device_input_vector, size);
    hipMalloc(&device_output_vector, size);

    hipMemcpy(device_input_vector, host_input_vector, size, hipMemcpyHostToDevice);
    hipMemcpy(device_output_vector, host_output_vector, size, hipMemcpyHostToDevice);


    dim3 block_dimension(8, 8);
    dim3 blocks_per_grid(
        (N + block_dimension.x - 1) / block_dimension.x,
        (M + block_dimension.y - 1) / block_dimension.y
    );

    matrix_scale<<<blocks_per_grid, block_dimension>>>(device_input_vector, device_output_vector, scalar, N, M);


    hipMemcpy(host_output_vector, device_output_vector, size, hipMemcpyDeviceToHost);


    hipFree(device_input_vector); hipFree(device_output_vector);
    free(host_input_vector); free(host_output_vector);

    return 0;
}