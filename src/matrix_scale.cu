
#include <hip/hip_runtime.h>
#include <iostream>
#include <hiprand/hiprand.h>


__global__ void matrix_scale(float *input, float *output, float scale, int N, int M) {
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    if (row < N && col < M) {
        output[row * N + col] = input[row * N + col] * scale;
    }
}


int main() {

    int number = 1 << 8;
    int N = number, M = number;
    size_t size = N * M * sizeof(float);
    float scalar = 2.0f;

    float *host_input_vector, *host_output_vector, *device_input_vector, *device_output_vector;

    host_input_vector = (float*)malloc(size);
    host_output_vector = (float*)malloc(size);
    
    // Populate host vector
    hiprandGenerator_t generator;
    hiprandCreateGeneratorHost(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(generator, (unsigned long long)clock());
    hiprandGenerateUniform(generator, host_input_vector, N * M);

    hipMalloc(&device_input_vector, size);
    hipMalloc(&device_output_vector, size);

    hipMemcpy(device_input_vector, host_input_vector, size, hipMemcpyHostToDevice);
    hipMemcpy(device_output_vector, host_output_vector, size, hipMemcpyHostToDevice);


    dim3 block_dimension(8, 8);
    dim3 blocks_per_grid(
        (N + block_dimension.x - 1) / block_dimension.x,
        (M + block_dimension.y - 1) / block_dimension.y
    );

    matrix_scale<<<blocks_per_grid, block_dimension>>>(device_input_vector, device_output_vector, scalar, N, M);


    hipMemcpy(host_output_vector, device_output_vector, size, hipMemcpyDeviceToHost);


    hipFree(device_input_vector); hipFree(device_output_vector);
    free(host_input_vector); free(host_output_vector);

    return 0;
}