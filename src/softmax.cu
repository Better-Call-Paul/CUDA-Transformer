#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include <ctime>
#include <cstdlib>
#include <cmath>
#include <hiprand.h>

#define SHMEM_SIZE 256 

__inline__ __device__ 
float warp_reduce(volatile float* shmem_ptr, int thread_id) {
    if (thread_id < 32) {
        shmem_ptr[thread_id] += (thread_id + 32 < blockDim.x) ? shmem_ptr[thread_id + 32] : 0;
        shmem_ptr[thread_id] += (thread_id + 16 < blockDim.x) ? shmem_ptr[thread_id + 16] : 0;
        shmem_ptr[thread_id] += (thread_id + 8 < blockDim.x) ? shmem_ptr[thread_id + 8] : 0;
        shmem_ptr[thread_id] += (thread_id + 4 < blockDim.x) ? shmem_ptr[thread_id + 4] : 0;
        shmem_ptr[thread_id] += (thread_id + 2 < blockDim.x) ? shmem_ptr[thread_id + 2] : 0;
        shmem_ptr[thread_id] += (thread_id + 1 < blockDim.x) ? shmem_ptr[thread_id + 1] : 0;
    }
    return shmem_ptr[thread_id];
}

__global__ void softmax(float *input, float *output, int N, int M) {
    
    extern __shared__ float shared_mem[];
    int thread_index = threadIdx.x;
    int global_index = blockIdx.x * blockDim.x + threadIdx.x;

    // Only compute exponentials for valid (non-padded) values 
    if (thread_index < N * M) {
        shared_mem[thread_index] = exp(shared_mem[thread_index]);
    }
    else {
        shared_mem[thread_index] = 0.0f;
    }

    // Ensure all exponentials are computed
    __syncthreads();

    // warp_reduce

    // get the value in the 0th index 
    if (threadIdx.x == 0) {
        
    }
}

void check_softmax(float *input, float *output, float *computed_output, int N, int M, const float tolerance) {
    for (int i = 0; i < N; i++) {
        float sum = 0.0f;
        for (int j = 0; j < M; j++) {
            computed_output[i * M + j] = exp(input[i * M + j]);
            sum += computed_output[i * M + j];
        }
        for (int k = 0; k < M; k++) {
            computed_output[i * M + k] /= sum;
            assert(std::fabs(computed_output[i * M + k] - output[i * M + k]) < tolerance);
        }
    }
}

void handle_cuda_error(hipError_t error, const char* message) {
    if (error != hipSuccess) {
        std::cerr << message << ": " << hipGetErrorString(error) << "\n";
        exit(-1);
    }
}

int main() {
    const float tolerance = 1e-3;
    int number = 1 << 8;
    int N = number, M = number;
    size_t size = N * M * sizeof(float);

    float *host_input_vector, *host_output_vector, *device_input_vector, *device_output_vector, *iterative_output_vector;

    host_input_vector = (float*)malloc(size);
    host_output_vector = (float*)malloc(size);
    iterative_output_vector = (float*)malloc(size);
    
    // Populate host vectors
    hiprandGenerator_t generator;
    hiprandCreateGeneratorHost(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(generator, (unsigned long long)clock());
    hiprandGenerateUniform(generator, host_input_vector, N * M);

    hipMalloc(&device_input_vector, size);
    hipMalloc(&device_output_vector, size);

    hipMemcpy(device_input_vector, host_input_vector, size, hipMemcpyHostToDevice);
    hipMemcpy(device_output_vector, host_output_vector, size, hipMemcpyHostToDevice);


    dim3 block_dimensions(8, 8);

    dim3 threads(
        (N + block_dimensions.x - 1) / block_dimensions.x,
        (M + block_dimensions.y - 1) / block_dimensions.y
    );

    softmax<<<block_quantity, threads, SHMEM_SIZE * sizeof(float)>>>(device_input_vector, device_output_vector, N, M);
    
    handle_cuda_error(hipGetLastError(), "Kernel Failed");

    hipMemcpy(host_output_vector, device_output_vector, size, hipMemcpyDeviceToHost);

    check_softmax(host_input_vector, host_output_vector, iterative_output_vector, N, M, tolerance);

    hipFree(device_input_vector);
    hipFree(device_output_vector);

    return 0;
}