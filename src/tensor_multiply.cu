
#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>
#include <cstdlib>
#include <cassert>

#define SHMEM_SIZE 8 * 8 * 8 // shared memory size

void handle_cuda_error(hipError_t error, const char* message) {
    if (error != hipSuccess) {
        std::cerr << message << ": " << hipGetErrorString(error) << "\n";
        exit(-1);
    }
}

/*
 * Checks kernel multiplation for 2-D Matrices
*/
void check_multiply_error_2d(float *a, float *b, float *c, int N, const float tolerance) {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            float expected = 0.0f;
            for (int k = 0; k < N; k++) {
                int a_idx = i * N + k, b_idx = k * N + j;
                expected += a[a_idx] * b[b_idx];
            }
            int c_idx = i * N + j;
            assert(std::fabs(c[c_idx] - expected) < tolerance);
        }
    }
    std::cout << "Kernel Multiplication Valid" << "\n";
}

/*
 * 2d Multiplication
*/
__global__ void tensor_multiply_2d(float *a, float *b, float *c, int N, int M) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int column = blockIdx.x * blockDim.x + threadIdx.x;
    float temp_sum = 0.0f;

    if (row < N && column < M) {
        for (int i = 0; i < N; i++) {
            temp_sum += a[row * N + i] * b[N * i + column];
        }
        c[row * N + column] = temp_sum;
    }
}

/*
 * Optimized 2D tensor multiplication
 * assumes square matrixes of dimensions N x N
*/
__global__ void cache_tiled_multiply_2d(float *a, float *b, float *c, int N, int tile_size) {

    __shared__ float A[SHMEM_SIZE];
    __shared__ float B[SHMEM_SIZE];

    int row = blockIdx.y * tile_size + threadIdx.y;
    int col = blockIdx.x * tile_size + threadIdx.x;

    float temp_sum = 0.0f;

    for (int i = 0; i < (N / tile_size); i++) {
        /*
         * Index Calculations
         * For A:
            row * n : global row (loop-invariant)
            i * tile_size : new set of colums each iteration
            thread.Idx : index of the column within the set
         * For B:
            i * tile_size * n : next set of rows each iteration
            threadIdx.y * n : row within that set
            column : index of the global column (loop-invariant)
        */
        A[(threadIdx.y * tile_size) + threadIdx.x] = a[row * N + (i * tile_size + threadIdx.x)];
        B[(threadIdx.y * tile_size) + threadIdx.x] = b[(i * tile_size * N + threadIdx.y * N) + col];

        // Make sure its loaded
        __syncthreads();

        for (int j = 0; j < tile_size; j++) {
            temp_sum += A[(threadIdx.y * tile_size) + j] * B[(j * tile_size) + threadIdx.x];
        }

        // Ensure no overwrites on shared memory values;
        __syncthreads();

    }
    c[(row * N) + col] = temp_sum;

}

int main(int argc, char *argv[]) {

    const float tolerance = 1e-4; // determines tolerance for all checks
    int number = 1 << 8;
    srand(time(NULL));
    float *host_vector_a, *host_vector_b, *host_vector_c;
    float *device_vector_a, *device_vector_b, *device_vector_c;

    
    // 2-D Matrix Calculations
    int N = number, M = number;
    int tile_size = 8; // for cache tiled operations
    size_t size = N * M * sizeof(float); 
    
    host_vector_a = (float*)malloc(size);
    host_vector_b = (float*)malloc(size);
    host_vector_c = (float*)malloc(size);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {   
            int index = i * M + j;
            host_vector_a[index] = static_cast<float>(rand()) / RAND_MAX;
            host_vector_b[index] = static_cast<float>(rand()) / RAND_MAX;
        }
    }

    hipMalloc(&device_vector_a, size);
    hipMalloc(&device_vector_b, size);
    hipMalloc(&device_vector_c, size);

    hipMemcpy(device_vector_a, host_vector_a, size, hipMemcpyHostToDevice);
    hipMemcpy(device_vector_b, host_vector_b, size, hipMemcpyHostToDevice);
    hipMemcpy(device_vector_c, host_vector_c, size, hipMemcpyHostToDevice);

    dim3 block_dimensions(8, 8);
    dim3 block_quantity(
        (N + block_dimensions.x - 1) / block_dimensions.x,
        (M + block_dimensions.y - 1) / block_dimensions.y
    );
    
    //tensor_multiply_2d<<<block_quantity, block_dimensions>>>(device_vector_a, device_vector_b, device_vector_c, N, M); // Naive
    cache_tiled_multiply_2d<<<block_quantity, block_dimensions>>>(device_vector_a, device_vector_b, device_vector_c, N, tile_size); // Cache Tiled


    hipError_t error = hipGetLastError();

    handle_cuda_error(error, "Kernel Failed");

    hipDeviceSynchronize();

    hipMemcpy(host_vector_c, device_vector_c, size, hipMemcpyDeviceToHost);

    check_multiply_error_2d(host_vector_a, host_vector_b, host_vector_c, N, tolerance);

    //2-D tensor end 
    

    free(host_vector_a);
    free(host_vector_b);
    free(host_vector_c);

    hipFree(device_vector_a);
    hipFree(device_vector_b);
    hipFree(device_vector_c);

    return 0;
}
